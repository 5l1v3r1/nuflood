#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "compute_time_step.h"

template<int block_size>
__global__ void ComputeTimeStepKernel(double* max_speed, int n) {
	__shared__ double s_max[block_size];

	int tid = threadIdx.x;
	s_max[tid] = 0.0;

	for (int i = tid; i < n; i += block_size) {
		s_max[tid] = fmax(s_max[tid], max_speed[i]);
	}

	__syncthreads();

	if (block_size >= 512) {
		if (tid < 256) s_max[tid] = fmax(s_max[tid], s_max[tid+256]);
		__syncthreads();
	}

	if (block_size >= 256) {
		if (tid < 128) s_max[tid] = fmax(s_max[tid], s_max[tid+128]);
		__syncthreads();
	}

	if (block_size >= 128) {
		if (tid < 64) s_max[tid] = fmax(s_max[tid], s_max[tid+64]);
		__syncthreads();
	}

	if (tid < 32) {
		volatile double *s_mem = s_max;

		if (block_size >= 64) s_mem[tid] = fmax(s_mem[tid], s_mem[tid+32]);

		if (tid < 16) {
			if (block_size >= 32) s_mem[tid] = fmax(s_mem[tid], s_mem[tid+16]);
			if (block_size >= 16) s_mem[tid] = fmax(s_mem[tid], s_mem[tid+8]);
			if (block_size >= 8) s_mem[tid] = fmax(s_mem[tid], s_mem[tid+4]);
			if (block_size >= 4) s_mem[tid] = fmax(s_mem[tid], s_mem[tid+2]);
			if (block_size >= 2) s_mem[tid] = fmax(s_mem[tid], s_mem[tid+1]);
		}

		if (tid == 0) max_speed[0] = s_mem[0];
	}
}

double ComputeTimeStep(GpuRaster<double>* max_speed, double desingularization) {
	int block_size = 1;
	int num_elements = max_speed->gpu_grid_dim().x *
	                   max_speed->gpu_grid_dim().y;

	for (int k = 1; k <= 512; k *= 2) {
		block_size = (num_elements >= k) ? k : block_size;
	}

	size_t shared_mem_size = block_size*sizeof(double) * (block_size <= 32) ? 2 : 1;

	switch (block_size) {
		case 512:
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(ComputeTimeStepKernel<512>), hipFuncCachePreferShared);
			ComputeTimeStepKernel<512> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 256:
			ComputeTimeStepKernel<256> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 128:
			ComputeTimeStepKernel<128> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 64:
			ComputeTimeStepKernel<64> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 32:
			ComputeTimeStepKernel<32> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 16:
			ComputeTimeStepKernel<16> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 8:
			ComputeTimeStepKernel<8> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 4:
			ComputeTimeStepKernel<4> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 2:
			ComputeTimeStepKernel<2> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
		case 1:
			ComputeTimeStepKernel<1> <<< 1, block_size, shared_mem_size, 0 >>>
				(max_speed->gpu_array(), num_elements);
			break;
	}

	GpuErrChk(hipPeekAtLastError());
	GpuErrChk(hipDeviceSynchronize());

	double domain_max_speed;
	hipMemcpy(&domain_max_speed, max_speed->gpu_array(), sizeof(double), hipMemcpyDeviceToHost);
	return max_speed->cellsize_x() / fmax(fmax(4.0 * domain_max_speed, desingularization), 10.0);
}
